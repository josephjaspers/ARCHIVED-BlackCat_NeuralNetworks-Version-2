
#include <hip/hip_runtime.h>
////
////#include "BC_NN_Functions.h"
////
//#include "Tensor.h"
//	__global__
//	void sigmoid(float* x, unsigned sz) {
//		for (int i = 0; i < sz; ++i) {
//			x[i] = 1 / (1+ pow(2.71828, -x[i]));
//		}
//	}
//	__global__
//	void sigmoid_deriv(float* x, unsigned sz)  {
//		for (int i = 0; i < sz; ++i) {
//			x[i] *= (1 - x[i]);
//		}
//	}
////
//	void sigmoid(Tensor<float, GPU>& x) { sigmoid<<<10, 128>>>(x.data(), x.size());}
//		void sigmoid_deriv(Tensor<float, GPU>& x)  { sigmoid_deriv<<<10, 128>>>(x.data(), x.size());}
//
////
